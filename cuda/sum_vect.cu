#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>

__global__
void vect_add_kernel(int *a, int *b, int *c, int n){
	unsigned i = (blockDim.x*blockIdx.x)+threadIdx.x;

	if(i<n)	c[i] = a[i]+b[i];
}

void vect_add(int *a, int *b, int *c, unsigned n, unsigned thc){
	unsigned size = n*sizeof(int);
	int *d_a, *d_b, *d_c;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);	
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	hipMalloc((void **)&d_c, size);
	vect_add_kernel<<<(ceil(n/thc), thc)>>>(d_a, d_b, d_c, n);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	cudeFree(d_a);
	cudeFree(d_b);
	cudeFree(d_c);
}

void print(int *v, unsigned s){
	for(unsigned i=0; i<t; i++)	std::cout << v[i] << " ";
	std::cout << "\n";
}

int int main(int argc, char const *argv[]){
	unsigned n = 1000;
	unsigned thc = 256;

	int *h_a = new int[n];
	int *h_b = new int[n];
	int *h_c = new int[n];

	unsigned i;
	for(i=0; i<n; i++){
		h_a[i] = i;
		h_b[i] = i;
	}

	vect_add(h_a, h_b, h_c, n, thc);

	print(h_c, n)

	delete h_a;
	delete h_b;
	delete h_c;

	return 0;
}