
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>

void print(int *v, unsigned s){
	for(unsigned i=0; i<s; i++)	std::cout << v[i] << " ";
	std::cout << "\n";
}

__global__
void vect_add_kernel(int *a, int *b, int *c, int n){
	unsigned i = (blockDim.x*blockIdx.x)+threadIdx.x;

	if(i<n)	c[i] = a[i]+b[i];
}

void vect_add(int *a, int *b, int *c, unsigned n, unsigned thc){
	unsigned size = n*sizeof(int);
	int *d_a, *d_b, *d_c;

	//print(a,n);
	//print(b,n);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);	
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	hipMalloc((void **)&d_c, size);
	vect_add_kernel<<<ceil(n/thc), thc>>>(d_a, d_b, d_c, n);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

int main(int argc, char const *argv[]){
	unsigned n = 100;
	unsigned thc = 25;

	int *h_a = new int[n];
	int *h_b = new int[n];
	int *h_c = new int[n];

	unsigned i;
	for(i=0; i<n; i++){
		h_a[i] = i;
		h_b[i] = i;
		h_c[i] = 0;
	}

	vect_add(h_a, h_b, h_c, n, thc);

	print(h_c, n);

	delete h_a;
	delete h_b;
	delete h_c;

	return 0;
}